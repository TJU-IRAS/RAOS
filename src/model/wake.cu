#include "hip/hip_runtime.h"
/*
 * Wakes in the simulation environment
 *
 * Author: Roice (LUO Bing)
 * Date: 2016-03-08 create this file (RAOS)
 */

#include <stdio.h>
#include "model/robot.h"
#include "model/wake_rotor.h"
#include "model/error_cuda.h"
#include "model/plume.h" // for wake-induced velocity computation of plume puffs
#include "model/SimModel.h"
#include "model/environment.h"

#define PI 3.14159265358979323846

// storing cuda device properties, for the sheduling of parallel computing
static hipDeviceProp_t prop;

// data
VortexMarker_t *wake_markers; // on-host array containing all of the markers, for convenience mem copying with GPU
VortexMarker_t *dev_wake_markers; // on-device array ...
VortexMarker_t *dev_wake_markers_mediate; // on-device array ...
int *idx_end_marker_fila; // on-host array containing index of endpoint of all vortex filaments, for parallel computing
int *dev_idx_end_marker_fila; // on-device array ...
#if defined(WAKE_BC_FAR)
FarWakeState_t* far_wakes;
FarWakeState_t* dev_far_wakes; // on-device array containing the far wake states of rotors
#endif
// wind
typedef struct
{
    float v[3];
} Wake_FreeStreamVector_t;
Wake_FreeStreamVector_t *free_stream;
Wake_FreeStreamVector_t *dev_free_stream; // on-device ... free stream vector at robots' position
int *idx_end_marker_robot; // endpoint of vortices of robots
int *dev_idx_end_marker_robot;

// vel += biot savert induction from segement a-b to position p
__device__ float3
biot_savart_induction(VortexMarker_t
a,
VortexMarker_t b, float3
p,
float3 vel
)
{
float3 ap, bp, ab, ba, ind;
float dbnrm_ab, dbnrm_ap, dbnrm_bp; // |vector|^2
float cos_apab, cos_babp, db_sin_apab; // cos(ap,ab) cos(ba,bp) sin(ap,ab)^2
float db_h, scale; // h^2, |ind|

// if marker a or b is at p, pass
if ( a.pos[0]==p.
x &&a
.pos[1]==p.
y &&a
.pos[2]==p.z ||
b.pos[0]==p.
x &&b
.pos[1]==p.
y &&b
.pos[2]==p.z ||
a.pos[0]==b.pos[0] && a.pos[1]==b.pos[1] && a.pos[2]==b.pos[2]) {
return
vel;
}

// get vectors AP, BP and AB & BA  [12 FLOPS]
//  AP [3 FLOPS]
ap.
x = p.x - a.pos[0];
ap.
y = p.y - a.pos[1];
ap.
z = p.z - a.pos[2];
/* // __fsub_ru() costs equal to simply minus
ap.x = __fsub_ru(p.x, a.pos[0]);
ap.y = __fsub_ru(p.y, a.pos[1]);
ap.z = __fsub_ru(p.z, a.pos[2]);
*/
//  BP [3 FLOPS]
bp.
x = p.x - b.pos[0];
bp.
y = p.y - b.pos[1];
bp.
z = p.z - b.pos[2];
/*
bp.x = __fsub_ru(p.x, b.pos[0]);
bp.y = __fsub_ru(p.y, b.pos[1]);
bp.z = __fsub_ru(p.z, b.pos[2]);
*/
//  AB [3 FLOPS]
ab.
x = b.pos[0] - a.pos[0];
ab.
y = b.pos[1] - a.pos[1];
ab.
z = b.pos[2] - a.pos[2];
/*
ab.x = __fsub_ru(b.pos[0], a.pos[0]);
ab.y = __fsub_ru(b.pos[1], a.pos[1]);
ab.z = __fsub_ru(b.pos[2], a.pos[2]);
*/
//  BA [3 FLOPS]
ba.
x = a.pos[0] - b.pos[0];
ba.
y = a.pos[1] - b.pos[1];
ba.
z = a.pos[2] - b.pos[2];
/*
ba.x = __fsub_ru(a.pos[0], b.pos[0]);
ba.y = __fsub_ru(a.pos[1], b.pos[1]);
ba.z = __fsub_ru(a.pos[2], b.pos[2]);
*/

// cos(ap-ab) and cos(ba-bp) [31 FLOPS]
dbnrm_ab = ab.x * ab.x + ab.y * ab.y + ab.z * ab.z; // [5 FLOPS]
dbnrm_ap = ap.x * ap.x + ap.y * ap.y + ap.z * ap.z; // [5 FLOPS]
dbnrm_bp = bp.x * bp.x + bp.y * bp.y + bp.z * bp.z; // [5 FLOPS]
/* // using __powf(x,y) to calculate squares costs more than simply multiply
dbnrm_ab = __powf(ab.x,2) + __powf(ab.y,2) + __powf(ab.z,2);
dbnrm_ap = __powf(ap.x,2) + __powf(ap.y,2) + __powf(ap.z,2);
dbnrm_bp = __powf(bp.x,2) + __powf(bp.y,2) + __powf(bp.z,2);
*/
cos_apab = (ap.x * ab.x + ap.y * ab.y + ap.z * ab.z) * __frsqrt_rn(dbnrm_ap * dbnrm_ab); // [8 FLOPS]
cos_babp = (ba.x * bp.x + ba.y * bp.y + ba.z * bp.z) * __frsqrt_rn(dbnrm_ab * dbnrm_bp); // [8 FLOPS]

// h, perpendicular distance from P to AB [3 FLOPS]
db_sin_apab = 1 - cos_apab * cos_apab; //[2 FLOPS]
if (db_sin_apab <= 0.0f) // sometimes |cos_apab| will be slightly larger than 1.0f due to computation err
db_h = 0.0f;
else {
db_h = dbnrm_ap * db_sin_apab; // [1 FLOPS]
}

// strength
scale = a.Gamma * db_h * 10000.0f * __frsqrt_rn(__powf(a.r * 100, 4) + db_h * db_h * 10000.0f * 10000.0f) *
        (cos_apab + cos_babp) / (4 * PI);

// get induced velocity
ind.
x = ap.y * bp.z - ap.z * bp.y;
ind.
y = ap.z * bp.x - ap.x * bp.z;
ind.
z = ap.x * bp.y - ap.y * bp.x;
scale = scale * __frsqrt_rn(ind.x * ind.x + ind.y * ind.y + ind.z * ind.z);
ind.
x = scale * ind.x;
ind.
y = scale * ind.y;
ind.
z = scale * ind.z;

// add induced velocity [3 FLOPS]
vel.x += ind.
x;
vel.y += ind.
y;
vel.z += ind.
z;

return
vel;
}

__device__ float3
tile_calculation_vel_markers
(float3
pos,
float3 vel,
int tile,
int tile_size,
int row_sgmts, VortexMarker_t
* markers,
int *index_end,
int num_fila,
int num_markers
)
{
int i; // the i-th colum marker in this tile
int idx_m; // the idx_m -th colum marker of whole N markers
int idx_fila; // index of fila
int shared_mem_offset = (tile % row_sgmts) * tile_size;

bool isend; // the marker to be calculated is an end point or not

extern __shared__ VortexMarker_t
tile_markers[];

// if this tile is not the first tile, we will use the last tile's end
if (tile > 0)
{
isend = false;
for (
idx_fila = 0;
idx_fila<num_fila;
idx_fila++) {
if (
tile *tile_size
-1 == index_end[idx_fila]) {
isend = true;
break;
}
}
if (isend == false &&
tile *tile_size<num_markers
)
vel = biot_savart_induction(markers[tile * tile_size - 1], tile_markers[shared_mem_offset], pos, vel);
}

for (
i = 0;
i<tile_size-1; i++)
{// compule all colum markers in this tile
idx_m = tile * tile_size + i;
if (idx_m >= num_markers-1)
break;
else
{
isend = false;
// check whether this marker is end point or not
for (
idx_fila = 0;
idx_fila<num_fila;
idx_fila++) {
if (idx_m == index_end[idx_fila]) {
isend = true;
break;
}
}
if (isend == false) // if this marker is not an end point
vel = biot_savart_induction(tile_markers[shared_mem_offset + i], tile_markers[shared_mem_offset + i + 1], pos, vel);
}
}

return
vel;
}

/* calculate velocities of markers, running on GPU
Note:
    This routine uses one-dimensional blocks and threads
    row_sgms is the number of segments to parallelly compute a row
    blockDim must equal to tile_size*row_sgmts
    gridDim == (num_markers + tile_size - 1) / tile_size
 */
__global__ void CalculateVelofMarkers(VortexMarker_t * markers, int * idx_end, int
num_fila,
int num_markers,
int tile_size,
int row_sgmts, Wake_FreeStreamVector_t
* wind,
int *idx_end_robot,
int num_robot
)
{
extern __shared__ VortexMarker_t
tile_markers[];

int idx, i;
float3 pos; // position of vortex marker to calculate velocity in this thread
float3 vel = {0.0f, 0.0f, 0.0f}; // velocity of this marker
int row = threadIdx.x % tile_size + blockIdx.x * tile_size; // get row number (the index of marker to be evaluated)

// get the marker of which the velocity to be calculated
if (row<num_markers) {
pos.
x = markers[row].pos[0];
pos.
y = markers[row].pos[1];
pos.
z = markers[row].pos[2];
}

// compute each tile
i = 0;
while (true) // every thread can enter this loop
{
idx = threadIdx.x + blockDim.x * i;
if (idx<num_markers)
tile_markers[threadIdx.x] = markers[idx]; // copy markers to shared mem for tile calculation
__syncthreads(); // make sure the shared mem has been loaded
if (row<num_markers)
vel = tile_calculation_vel_markers(pos, vel, idx / tile_size, tile_size, row_sgmts, markers, idx_end, num_fila,
                                   num_markers);

__syncthreads(); // make sure every thread has done the calc of this tile

if (blockDim.
x *(i
+1) < num_markers)
i++;
else
break;
}

// save the result of this segment to the end of the shared mem
//  at present the shared memory is of no use for computation, so it can be used to temporarily store the segment results
if (row<num_markers) {
tile_markers[threadIdx.x].vel[0] = vel.
x;
tile_markers[threadIdx.x].vel[1] = vel.
y;
tile_markers[threadIdx.x].vel[2] = vel.
z;
}

__syncthreads(); // make sure the segment results are stored

if (threadIdx.x<tile_size && row < num_markers) {
// sum the velocities computed by multiple (num of row_sgms) threads
vel.
x = 0.0f;
vel.
y = 0.0f;
vel.
z = 0.0f;
for (
i = 0;
i<row_sgmts;
i++) {
vel.x += tile_markers[threadIdx.x%tile_size+
i *tile_size
].vel[0];
vel.y += tile_markers[threadIdx.x%tile_size+
i *tile_size
].vel[1];
vel.z += tile_markers[threadIdx.x%tile_size+
i *tile_size
].vel[2];
}
// Save the result in global memory for the integration step.
/*TODO:*/
for (
i = 0;
i<num_robot;
i++) {
if (row <= idx_end_robot[i]) {
markers[row].vel[0] = vel.x + wind[i].v[0];
markers[row].vel[1] = vel.y + wind[i].v[1];
markers[row].vel[2] = vel.z + wind[i].v[2];
break;
}
}
}
}

/* calculate new positions of markers, running on GPU 
    Forward Euler  
 */
__global__ void CalculatePosofMarkersFE(VortexMarker_t * markers, int
num_markers,
float dt
)
{
VortexMarker_t mkr;
int tid = threadIdx.x + blockIdx.x * blockDim.x;

if (tid<num_markers) {
mkr = markers[tid];

for (
int i = 0;
i < 3; i++)
mkr.pos[i] += mkr.vel[i]*
dt;
}
if (tid<num_markers)
markers[tid] =
mkr;
}

/* average velocity of two sequence of markers
   the number of markers should be the same
   This routine is used for the correction step of predictor-corrector integration
   scheme of FVM, which involves the average computation of previous velocity and
   predicted velocity

   the vel of array markers and markers_temp will be averaged and stored in markers
 */
__global__ void AverageVelofMarkers(VortexMarker_t * markers, VortexMarker_t * markers_temp, int
num_markers)
{
VortexMarker_t mkr, mkr_temp;
int tid = threadIdx.x + blockIdx.x * blockDim.x;

if (tid<num_markers) {
mkr = markers[tid];
mkr_temp = markers_temp[tid];

for (
int i = 0;
i < 3; i++)
mkr.vel[i] = 0.5*(mkr.vel[i] + mkr_temp.vel[i]);
}
if (tid<num_markers)
markers[tid] =
mkr;
}

/* calculate vortex core radius of markers */
__global__ void CalculateVtxCoreofMarkers(VortexMarker_t * markers, int
num_markers,
int *idx_end,
int num_fila,
float dt
)
{
VortexMarker_t mkr;
float t; // time t, second
int fila_start = 0, fila_end = idx_end[num_fila - 1]; // start and end index of a filament in markers
int tid = threadIdx.x + blockIdx.x * blockDim.x;

if (tid<num_markers) {
mkr = markers[tid];

// calculate life time of this marker (vortex segment)
for (
int i = 0;
i<num_fila;
i++) {
if (idx_end[i] < tid) { // find which filament this marker belongs to
if (idx_end[i] > fila_start)
fila_start = idx_end[i];
}
else {
if (idx_end[i] < fila_end)
fila_end = idx_end[i];
}
}
t = (fila_end - tid) * dt;
/* r_c(t) = sqrt( r_init^2 + 4*alpha*delta*nu*t )
   Here we assume r_init is 0.005 m
   alpha is Lamb's constant which is 1.25643
   delta is is a function of vortex Reynolds number, delta > 1 (laminar when delta = 1)
   here we assume delta = 8
   nu is viscous constant of air at 25 degree temperature: 0.01834
*/
mkr.
r = __fsqrt_ru(mkr.r_init * mkr.r_init + 4.0f * 1.25643f * 4.0 * 0.01834 * t);
}
if (tid<num_markers)
markers[tid] =
mkr;
}

__device__ float complete_elliptic_int_first(float k)
{
    return PI / 2.0 * (1.0 + 0.5 * 0.5 * k * k + 0.5 * 0.5 * 0.75 * 0.75 * __powf(k, 4));
}

__device__ float complete_elliptic_int_second(float k)
{
    return PI / 2.0 * (1.0 - 0.5 * 0.5 * k * k - 0.5 * 0.5 * 0.75 * 0.75 * __powf(k, 4) / 3.0);
}

__device__ void
induced_velocity_vortex_ring(float *center, float radius, float Gamma, float core_radius, VortexMarker_t *mkr)
{
    float op[3], op_z, op_r, db_op_z, db_op_r;
    float u_z, u_r, m, a, b;
    float db_radius, db_delta;

    db_radius = radius * radius;
    db_delta = core_radius * core_radius;
    // vector op
    for (int j = 0; j < 3; j++)
        op[j] = mkr->pos[j] - center[j];
    if (op[0] == 0.0f && op[1] == 0.0f && op[2] == 0.0f) // P is at center
        mkr->vel[2] += Gamma / (2 * radius);
    else
    {
        // op_z, cylindrical coord
        op_z = op[2]; // implies ring direction of (0, 0, 1)
        db_op_z = op_z * op_z;
        // op_r, cylindrical coord
        db_op_r = op[0] * op[0] + op[1] * op[1] + op[2] * op[2] - op_z * op_z;
        op_r = __fsqrt_ru(db_op_r);
        // a, A, m
        a = __fsqrt_ru((op_r + radius) * (op_r + radius) + db_op_z + db_delta);
        b = (op_r - radius) * (op_r - radius) + db_op_z + db_delta;
        m = 4 * op_r * radius / (a * a);
        // u_z, cylindrical coord 
        u_z = Gamma / (2 * PI * a) * ((-(db_op_r - db_radius + db_op_z + db_delta) / b)
                                      * complete_elliptic_int_second(m) + complete_elliptic_int_first(m));
        // u_r, cylindrical coord
        u_r = Gamma * op_z / (2 * PI * op_r * a) * (((db_op_r + db_radius + db_op_z + db_delta) / b)
                                                    * complete_elliptic_int_second(m) - complete_elliptic_int_first(m));
        // map u_z, u_r to cartesian coord
        mkr->vel[2] += u_z; // downward
        mkr->vel[0] += op[0] * u_r / __fsqrt_ru(op[0] * op[0] + op[1] * op[1]);
        mkr->vel[1] += op[1] * u_r / __fsqrt_ru(op[0] * op[0] + op[1] * op[1]);
    }
}

#if defined(WAKE_BC_FAR)
/* calculate & add induced velocity of far wake BC */
__global__ void CalculateIndVelofFarWakeBC(VortexMarker_t* markers, int num_markers, FarWakeState_t* rings, int num_rings)
{
    VortexMarker_t mkr;
    float radius, Gamma; // radius and strength of a vortex ring 
    float center[3]; // center of the vortex ring
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < num_markers) {
        mkr = markers[tid];
        /* calculate induced velocity of far wake vortex ring */
        for (int i = 0; i < num_rings; i++) {
            if (rings[i].initialized == true) {
                radius = rings[i].radius;
                Gamma = __fsqrt_ru(rings[i].Gamma*rings[i].Gamma); // Gamma>=0
                center[0] = rings[i].center[0];
                center[1] = rings[i].center[1];
                for (int j = 0; j < 1; j++) {
                    center[2] = rings[i].center[2] + j*rings[i].gap;
                    induced_velocity_vortex_ring(&center[0], radius, Gamma, rings[i].core_radius, &mkr);
                }
            }
        }
    }
    if (tid < num_markers)
        markers[tid] = mkr;
}
#endif

/* update all of the rotor wakes in the environment
 * this routine will traverse all of the robot instances
 * and update all of the Lagrangian markers for all robots */
void WakesUpdate(std::vector<Robot *> *robots, const char *integration_scheme, SimState_t *sim_state,
                 SimEnvInfo *sim_env_info)
{
    unsigned int idx_robot, idx_rotor;
    int idx_blade;
    int num_blade = 0;
    int addr_cp_markers = 0; // index for copy marker states from rotor wake to wake_markers
#if defined(WAKE_BC_FAR)
    int addr_cp_rotors = 0;
#endif

/* Step 1: update velocity & position of markers */

    // Phase 1: collect all vortex markers to a memory, for GPU computing
    //  the markers are placed contiguously, fila to fila
    for (idx_robot = 0; idx_robot < robots->size(); idx_robot++)
    {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++)
        {
#if defined(WAKE_BC_FAR)
            memcpy(&far_wakes[addr_cp_rotors++], &(robots->at(idx_robot)->wakes.at(idx_rotor)->far_wake), sizeof(FarWakeState_t));
#endif
            for (idx_blade = 0;
                 idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++)
            {
                std::copy(robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->begin(),
                          robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->end(),
                          &wake_markers[addr_cp_markers]);
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size();
                idx_end_marker_fila[num_blade] = addr_cp_markers - 1; // the address of the last element, hence -1
                num_blade++;
            }
        }
        idx_end_marker_robot[idx_robot] = addr_cp_markers - 1; // the address of the last element of a robot
    }// traversed all rotor wakes and got total number of markers
#if defined(WAKE_IGE)
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            for (idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++) {
                std::copy(robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade+robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades]->begin(),
                    robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade+robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades]->end(),
                    &wake_markers[addr_cp_markers]);
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade+robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades]->size();
                idx_end_marker_fila[num_blade] = addr_cp_markers-1; // the address of the last element, hence -1
                num_blade++;
            }
        }
    }
#endif
    // get wind vectors
    for (unsigned int i = 0; i < robots->size(); i++)
        sim_env_info->measure_wind(robots->at(i)->state.pos, free_stream[i].v);

    // Phase 2: copy array wake_markers & idx_wake_markers to GPU's dev_wake_markers
    HANDLE_ERROR(hipMemcpy(dev_wake_markers, wake_markers,
                            addr_cp_markers * sizeof(VortexMarker_t), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_idx_end_marker_fila, idx_end_marker_fila,
                            num_blade * sizeof(int), hipMemcpyHostToDevice));
#if defined(WAKE_BC_FAR)
    HANDLE_ERROR( hipMemcpy(dev_far_wakes, far_wakes,
                addr_cp_rotors*sizeof(FarWakeState_t), hipMemcpyHostToDevice) );
#endif
    HANDLE_ERROR(hipMemcpy(dev_free_stream, free_stream,
                            robots->size() * sizeof(Wake_FreeStreamVector_t), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_idx_end_marker_robot, idx_end_marker_robot,
                            robots->size() * sizeof(int), hipMemcpyHostToDevice));

    // Phase 3: parallel computing
    //  determine threads per block and blocks number, at present addr_cp_markers contains total num of markers
    int p, q, threads, blocks;
    p = prop.warpSize; // tile size
    q = 4; // number of segments of a row
    threads = std::min(p * q, prop.maxThreadsPerBlock);

    //  launch gpu computing
    //   the end address of shared memory contains the end element of 'last' tile for 'next' tile computation
    hipError_t err;

    //  <1> calculate velocity of Lagrangian markers, n-1 state
    blocks = (addr_cp_markers + p - 1) / p;
    //  Note: here omitted checks for max number of blocks, since in RAO problem the vortex markers
    //        rarely exceeds 65535*threads.
    CalculateVelofMarkers << < blocks, threads, (threads) * sizeof(VortexMarker_t) >> >
                                                (dev_wake_markers, dev_idx_end_marker_fila, num_blade, addr_cp_markers, p, q, dev_free_stream, dev_idx_end_marker_robot, robots->size());
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    //      add far wake boundary condition effect
#if defined(WAKE_BC_FAR)
    blocks = (addr_cp_markers + threads -1)/threads;
    CalculateIndVelofFarWakeBC<<<blocks, threads>>>(dev_wake_markers, addr_cp_markers, dev_far_wakes, addr_cp_rotors);
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
#endif

    //  <2> calculate position of Lagrangian markers, predict
    blocks = (addr_cp_markers + threads - 1) / threads;
    CalculatePosofMarkersFE << < blocks, threads >> > (dev_wake_markers, addr_cp_markers, sim_state->dt);
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

    if (strcmp(integration_scheme, "PC") == 0) // predictor-corrector
    {
        //  <3> calculate mediate velocity of predicted positions for corrector
        HANDLE_ERROR(hipMemcpy(dev_wake_markers_mediate, dev_wake_markers,
                                addr_cp_markers * sizeof(VortexMarker_t), hipMemcpyDeviceToDevice));
        blocks = (addr_cp_markers + p - 1) / p;
        CalculateVelofMarkers << < blocks, threads, (threads) * sizeof(VortexMarker_t) >> >
                                                    (dev_wake_markers_mediate, dev_idx_end_marker_fila, num_blade, addr_cp_markers, p, q, dev_free_stream, dev_idx_end_marker_robot, robots->size());
        err = hipGetLastError();
        if (err != hipSuccess)
            printf("Error: %s\n", hipGetErrorString(err));
        //  <4> calculate final position of markers, correct
        blocks = (addr_cp_markers + threads - 1) / threads;
        AverageVelofMarkers << < blocks, threads >> > (dev_wake_markers, dev_wake_markers_mediate, addr_cp_markers);
        err = hipGetLastError();
        if (err != hipSuccess)
            printf("Error: %s\n", hipGetErrorString(err));
        CalculatePosofMarkersFE << < blocks, threads >> > (dev_wake_markers, addr_cp_markers, sim_state->dt);
        err = hipGetLastError();
        if (err != hipSuccess)
            printf("Error: %s\n", hipGetErrorString(err));
    }
    //  <5> calculate vortex core growth
    //    the argument num_blade, num_markers and dt are used to compute lifetime
    blocks = (addr_cp_markers + threads - 1) / threads;
    CalculateVtxCoreofMarkers << < blocks, threads >> >
                                           (dev_wake_markers, addr_cp_markers, dev_idx_end_marker_fila, num_blade,
                                               20.0 / 360.0 / 50.0);
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

    // Phase 4: retrieve data dev_wake_markers from GPU to wake_markers
    HANDLE_ERROR(hipMemcpy(wake_markers, dev_wake_markers,
                            addr_cp_markers * sizeof(VortexMarker_t), hipMemcpyDeviceToHost));

    // Phase 5: GPU computed, distribute the memory to every rotor wake states
    addr_cp_markers = 0;
    for (idx_robot = 0; idx_robot < robots->size(); idx_robot++)
    {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++)
        {
            for (idx_blade = 0;
                 idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++)
            {
                std::copy(&wake_markers[addr_cp_markers],
                          &wake_markers[addr_cp_markers +
                                        robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size()],
                          robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->data());
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size();
            }
        }
    }
#if defined(WAKE_IGE)
    for(idx_robot = 0; idx_robot < robots->size(); idx_robot++) {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++) {
            for (idx_blade = 0; idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++) {
                std::copy(&wake_markers[addr_cp_markers],
                    &wake_markers[addr_cp_markers+robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade+robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades]->size()],
                robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade+robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades]->data());
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade+robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades]->size();
            }
        }
    }
#endif


/* Step 2: maintain markers of wakes */
    for (idx_robot = 0; idx_robot < robots->size(); idx_robot++)
    {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++)
        {
            if (sim_state->wake_initialized)
                robots->at(idx_robot)->wakes.at(idx_rotor)->maintain("turn_by_turn");
            else
                robots->at(idx_robot)->wakes.at(idx_rotor)->maintain("one_by_one");
        }
    }
}

/* init parallelization of wakes computation */
void WakesInit(std::vector<Robot *> *robots)
{
    /* print the properties of all the graphic cards this machine has */
    int count; // number of devices
    HANDLE_ERROR(hipGetDeviceCount(&count));
    for (int i = 0; i < count; i++)
    {// print out info of all graphic cards
        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        printf("======== Card %d ========\n", i + 1);
        printf("Graphic card name: %s\n", prop.name);
        printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("Total global memory: %ld MByte\n", prop.totalGlobalMem / 1024 / 1024);
        printf("Total constant memoty: %ld kByte\n", prop.totalConstMem / 1024);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp: %d\n", prop.regsPerBlock);
        printf("Threads in warp: %d\n", prop.warpSize);
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1],
               prop.maxThreadsDim[2]);
        printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("\n");
    }
    if (count > 1)
    {// multiple graphic cards
        if (count == 2)
        {
            HANDLE_ERROR(hipSetDevice(1));   //Set the second graphic to calculate
        } else
        {
            printf(
                "Warning: Multiple graphic cards have been found on this machine. Please modify the function WakeInit in the file src/model/wake.cu to choose the most appropriate card.\n");
            exit(EXIT_FAILURE); // force the user to choose which card to use
        }
    } else if (count <= 0)
    {// no graphic card found
        printf(
            "Error: No graphic cards have been found on this machine. Please run this program on the machine with NVIDIA graphic cards.\n");
        exit(EXIT_FAILURE);
    }

    /* Init GPU computation */
    // Note: the number of rotor wakes & vortex filaments are fixed since simulation starts

    // traverse all rotor wakes and got total max number of markers and fila, for allocating mem
    int max_num_markers = 0;
    int max_num_rotors = 0;
    int max_num_fila = 0;
    for (unsigned int idx_robot = 0; idx_robot < robots->size(); idx_robot++)
    {// traverse all robots
        for (unsigned int idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++)
        {// traverse all rotors
            max_num_rotors++;
            for (int idx_blade = 0;
                 idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++)
            {// traverse all blades
#if defined(WAKE_IGE)
                max_num_fila+=2;
#else
                max_num_fila++;
#endif
#if defined(WAKE_IGE)
                max_num_markers += 2*robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->capacity();
#else
                max_num_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->capacity();
#endif
            }
        }
    }

    // allocate a page-locked host memory containing all of the marker states
    HANDLE_ERROR(hipHostAlloc((void **) &wake_markers,
                               max_num_markers * sizeof(*wake_markers), hipHostMallocDefault));
    // allocate device memory as big as the host's
    HANDLE_ERROR(hipMalloc((void **) &dev_wake_markers,
                            max_num_markers * sizeof(*dev_wake_markers)));
    HANDLE_ERROR(hipMalloc((void **) &dev_wake_markers_mediate,
                            max_num_markers * sizeof(*dev_wake_markers_mediate)));

    // allocate host memory containing the index of wake_markers
    HANDLE_ERROR(hipHostAlloc((void **) &idx_end_marker_fila,
                               max_num_fila * sizeof(*idx_end_marker_fila), hipHostMallocDefault));
    // allocate device memory containing the indexes
    HANDLE_ERROR(hipMalloc((void **) &dev_idx_end_marker_fila,
                            max_num_fila * sizeof(*dev_idx_end_marker_fila)));

    // allocate host memory containing wind vectors at robots' positions
    HANDLE_ERROR(hipHostAlloc((void **) &free_stream,
                               robots->size() * sizeof(Wake_FreeStreamVector_t), hipHostMallocDefault));
    // allocate device memory
    HANDLE_ERROR(hipMalloc((void **) &dev_free_stream,
                            robots->size() * sizeof(Wake_FreeStreamVector_t)));

    // allocate host memory containing index to identify which robot the vortices belong to
    HANDLE_ERROR(hipHostAlloc((void **) &idx_end_marker_robot,
                               robots->size() * sizeof(*idx_end_marker_robot), hipHostMallocDefault));
    // allocate device memory
    HANDLE_ERROR(hipMalloc((void **) &dev_idx_end_marker_robot,
                            robots->size() * sizeof(*dev_idx_end_marker_robot)));

#if defined(WAKE_BC_FAR)
    // allocate host memory containing the far wake states
    HANDLE_ERROR( hipHostAlloc((void**)&far_wakes,
        max_num_rotors*sizeof(*far_wakes), hipHostMallocDefault) );
    // allocate device memory containing the far wake states
    HANDLE_ERROR( hipMalloc((void**)&dev_far_wakes,
        max_num_rotors*sizeof(*dev_far_wakes)) );
#endif
}

/* close GPU computation */
void WakesFinish(void)
{
    // free device memory
    HANDLE_ERROR(hipFree(dev_idx_end_marker_fila));
    HANDLE_ERROR(hipFree(dev_wake_markers));
    HANDLE_ERROR(hipFree(dev_free_stream));
    HANDLE_ERROR(hipFree(dev_idx_end_marker_robot));
#if defined(WAKE_BC_FAR)
    HANDLE_ERROR( hipFree(dev_far_wakes) );
#endif
    // free host memory
    HANDLE_ERROR(hipHostFree(idx_end_marker_fila));
    HANDLE_ERROR(hipHostFree(wake_markers));
    HANDLE_ERROR(hipHostFree(free_stream));
    HANDLE_ERROR(hipHostFree(idx_end_marker_robot));
#if defined(WAKE_BC_FAR)
    HANDLE_ERROR( hipHostFree(far_wakes) );
#endif
}

/*************************************************************************/
/*************** Calculate Induced Velocity at Plume puffs ***************/
/*************************************************************************/
// these functions can be called after WakesInit

FilaState_t *plume_puffs; // on-host ...
FilaState_t *dev_plume_puffs; // on-device array containing the states of plume puffs

__global__ void CalculateIndVelatPlumePuffs(VortexMarker_t * markers, int * idx_end, int
num_fila,
int num_markers,
int tile_size,
int row_sgmts, FilaState_t
* plume,
int num_puffs
)
{
extern __shared__ VortexMarker_t
tile_markers[];
float3 pos; // position of plume puff to calculate velocity in this thread
float3 vel = {0.0f, 0.0f, 0.0f}; // velocity of this marker
int idx, i;
int row = threadIdx.x % tile_size + blockIdx.x * tile_size; // get row number (the index of puff to be evaluated)

// get the plume fila (puff) which the velocity to be calculated
if (row<num_puffs)
{
pos.
x = plume[row].pos[0];
pos.
y = plume[row].pos[1];
pos.
z = plume[row].pos[2];
}

// compute each tile
i = 0;
while (true) // every thread can enter this loop
{
idx = threadIdx.x + blockDim.x * i;
if (idx<num_markers)
tile_markers[threadIdx.x] = markers[idx]; // copy markers to shared mem for tile calculation
__syncthreads(); // make sure the shared mem has been loaded
if (row<num_puffs)
vel = tile_calculation_vel_markers(pos, vel, idx / tile_size, tile_size, row_sgmts, markers, idx_end, num_fila,
                                   num_markers);

__syncthreads(); // make sure every thread has done the calc of this tile

if (blockDim.
x *(i
+1) < num_markers)
i++;
else
break;
}

// save the result of this segment to the end of the shared mem
//  at present the shared memory is of no use for computation, so it can be used to temporarily store the segment results
if (row<num_puffs) {
tile_markers[threadIdx.x].vel[0] = vel.
x;
tile_markers[threadIdx.x].vel[1] = vel.
y;
tile_markers[threadIdx.x].vel[2] = vel.
z;
}

__syncthreads(); // make sure the segment results are stored

if (threadIdx.x<tile_size && row < num_puffs) {
// sum the velocities computed by multiple (num of row_sgms) threads
vel.
x = 0.0f;
vel.
y = 0.0f;
vel.
z = 0.0f;
for (
i = 0;
i<row_sgmts;
i++) {
vel.x += tile_markers[threadIdx.x%tile_size+
i *tile_size
].vel[0];
vel.y += tile_markers[threadIdx.x%tile_size+
i *tile_size
].vel[1];
vel.z += tile_markers[threadIdx.x%tile_size+
i *tile_size
].vel[2];
}
// Save the result in global memory for the integration step.
plume[row].vel[0] = vel.
x;
plume[row].vel[1] = vel.
y;
plume[row].vel[2] = vel.
z;
}
}

void WakesIndVelatPlumePuffsUpdate(std::vector<Robot *> *robots, std::vector<FilaState_t> *plume)
{
    unsigned int idx_robot, idx_rotor;
    int idx_blade;
    int addr_cp_markers = 0, num_blade = 0;
    // Step 1: collect all vortex markers & puffs to one memory, respectively, for GPU computing
    //  the markers are placed contiguously, fila to fila
    for (idx_robot = 0; idx_robot < robots->size(); idx_robot++)
    {
        for (idx_rotor = 0; idx_rotor < robots->at(idx_robot)->wakes.size(); idx_rotor++)
        {
            for (idx_blade = 0;
                 idx_blade < robots->at(idx_robot)->wakes.at(idx_rotor)->rotor_state.frame.n_blades; idx_blade++)
            {
                std::copy(robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->begin(),
                          robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->end(),
                          &wake_markers[addr_cp_markers]);
                addr_cp_markers += robots->at(idx_robot)->wakes.at(idx_rotor)->wake_state[idx_blade]->size();
                idx_end_marker_fila[num_blade] = addr_cp_markers - 1; // the address of the last element, hence -1
                num_blade++;
            }
        }
    }// traversed all rotor wakes and got total number of markers
    //  collect puffs to a memory
    std::copy(plume->begin(), plume->end(), &plume_puffs[0]);

    // Step 2: copy array wake_markers, idx_wake_markers, plume_puffs to GPU's version
    HANDLE_ERROR(hipMemcpy(dev_wake_markers, wake_markers,
                            addr_cp_markers * sizeof(VortexMarker_t), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_idx_end_marker_fila, idx_end_marker_fila,
                            num_blade * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_plume_puffs, plume_puffs,
                            plume->size() * sizeof(FilaState_t), hipMemcpyHostToDevice));

    // Step 3: compute induced velocity of plume puffs
    //  determine threads per block and blocks number, at present addr_cp_markers contains total num of markers
    int p, q, threads, blocks;
    p = prop.warpSize; // tile size
    q = 4; // number of segments of a row
    threads = std::min(p * q, prop.maxThreadsPerBlock);
    //  launch gpu computing
    hipError_t err;
    //  <1> calculate velocity of Lagrangian markers, n-1 state
    blocks = (plume->size() + p - 1) / p;
    //  Note: here omitted checks for max number of blocks, since in RAO problem the vortex markers
    //        rarely exceeds 65535*threads.
    CalculateIndVelatPlumePuffs << < blocks, threads, threads * sizeof(VortexMarker_t) >> >
                                                      (dev_wake_markers, dev_idx_end_marker_fila, num_blade, addr_cp_markers, p, q, dev_plume_puffs, plume->size());
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

    // Step 4: distribute puffs to host and give back
    HANDLE_ERROR(hipMemcpy(plume_puffs, dev_plume_puffs,
                            plume->size() * sizeof(FilaState_t), hipMemcpyDeviceToHost));
    std::copy(&plume_puffs[0], &plume_puffs[plume->size()], plume->data());
}

void WakesIndVelatPlumePuffsInit(std::vector<Robot *> *robots, std::vector<FilaState_t> *plume)
{
    // allocate a page-locked host memory containing all of the plume puffs states
    HANDLE_ERROR(hipHostAlloc((void **) &plume_puffs,
                               MAX_NUM_PUFFS * sizeof(*plume_puffs), hipHostMallocDefault));
    // allocate device memory as big as the host's
    HANDLE_ERROR(hipMalloc((void **) &dev_plume_puffs,
                            MAX_NUM_PUFFS * sizeof(*dev_plume_puffs)));
}

/* End of file wake.cu */
